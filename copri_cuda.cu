
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <stdbool.h>
#include <assert.h>
#include <limits.h>

#define CHECK(call)															\
{																			\
	const hipError_t error = call;											\
	if (error != hipSuccess) {												\
		printf("Error: %s:%d ",__FILE__,__LINE__);							\
		printf("code: %d, reason: %s\n",error,hipGetErrorString(error));	\
		exit(1);															\
	}																		\
}

// ***********************************************************************
// Measuring time
// ***********************************************************************

double timestamp ( clockid_t clk_id ) {
	struct timespec t;
	int ret = clock_gettime(clk_id,&t);
	assert(ret == 0);
	double ts = ((double) t.tv_sec) + ((double) t.tv_nsec) / 10e9;
	return ts; // In seconds
}

double now () {
	return timestamp(CLOCK_REALTIME);
}

double clock_resolution_ns ( clockid_t clk_id ) {
	struct timespec t;
	int ret = clock_getres(clk_id,&t);
	assert(ret == 0);
	assert(t.tv_sec == 0);
	return (double) t.tv_nsec;
}

// ***********************************************************************
// CUDA kernel to test candidate c for prime number
// To decease possible congestion while incrementing the number of
// primes found, an array of counters indexed by the threadIdx.x
// is used. Still, threads with identical id but in different blocks
// might run in parallel if the GPU has multiple processors (28 in
// my case), so an atomic add operation is needed.
// ***********************************************************************

__global__ void prime_test( int *counter ) {
	int c = (blockIdx.x * blockDim.x + threadIdx.x) * 2 + 3;
	bool a_prime = true;
	for (int d = 3; d*d <= c; d += 2) {
		if (c % d == 0) {
			a_prime = false;
			break;
		}
	}
	if (a_prime) {
		atomicAdd(&(counter[threadIdx.x]), 1);
	}
}

// ***********************************************************************
// Count all primes from 3 up to limit using the streaming capabilities
// of an CUDA-capable GPU
// ***********************************************************************

int main(int ac, char **av) {
		if (sizeof(int) != 4) {
		fprintf(stderr,"sizeof(int) is not 4 (%lu instead)\n",sizeof(int));
		exit(-1);
	}
	if (ac != 2) {
		fprintf(stderr,"usage: %s <limit> - count primes in [1,limit); limit==-1: MAX_INT-1\n",av[0]);
		exit(-1);
	}
	int limit = atoi(av[1]);
	if (limit == -1)
		limit = INT_MAX-1;


	CHECK(hipSetDevice(0));
	hipDeviceProp_t deviceProp;
	CHECK(hipGetDeviceProperties(&deviceProp, 0));
	// printf("CUDA device warp size is %d\n", deviceProp.warpSize);
	// printf("CUDA device max threads per block is %d\n", deviceProp.maxThreadsPerBlock);
	// printf("CUDA device max grid size is %d\n", deviceProp.maxGridSize[0]);

	int block_size = 1000;
	assert(block_size < deviceProp.maxThreadsPerBlock);
	int grid_size = limit / block_size / 2;
	assert(grid_size < deviceProp.maxGridSize[0]);

	// printf("block size is %d and grid size (number of blocks) is %d\n", block_size, grid_size);

	double start = now();

	int *counter = (int *) malloc(block_size * sizeof(int));
	for (int i = 0; i < block_size; i++)
		counter[i] = 0;

	int *dev_counter = 0;
	CHECK(hipMalloc((void**) &dev_counter, block_size * sizeof(int)));
	CHECK(hipMemcpy(dev_counter, counter, block_size * sizeof(int), hipMemcpyHostToDevice));

	prime_test <<<grid_size,block_size>>> (dev_counter);
	CHECK(hipDeviceSynchronize());

	int n_primes = 1;
	CHECK(hipMemcpy(counter, dev_counter, block_size * sizeof(int), hipMemcpyDeviceToHost));
	for (int i = 0; i < block_size; i++)
		n_primes += counter[i];

	double stop = now();
	double exectime = stop - start;

	printf("%s, limit=%d, n_primes=%d, time(s)=<%e>\n",av[0],limit,n_primes,exectime);

	hipFree(dev_counter);
	free(counter);

	hipDeviceReset();
}
